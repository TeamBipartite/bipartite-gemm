/**
 * Driver for the benchmark comparison. Generates random data,
 * runs the CPU baseline, and then runs your code.
 */

#include <chrono>   // for timing
#include <iostream> // std::cout, std::endl
#include <iterator> // std::ostream_iterator
#include <vector>

#ifndef NO_OPENBLAS
#include <cblas.h>
#endif

#include "dense_graph.h"
#include "sparse_graph.h"

#include "data_generator.h"
#include "data_types.h"

/** 
 * Pads the given n to the least multiple of 32 not less than n
 */
constexpr std::size_t get_padded_sz(std::size_t n)
{
    return n%32 ? n + (32 - n%32) : n;
}

/**
 * Runs timing tests on a CUDA graph implementation.
 * Consists of independently constructing the graph and then
 * modifying it to its two-hop neighbourhood.
 */
template < typename DeviceGraph >
void run( DeviceGraph *g, csc485b::a2::edge_t const * d_edges, std::size_t m, std::size_t n )
{
    hipDeviceSynchronize();
    auto const build_start = std::chrono::high_resolution_clock::now();

    // this code doesn't work yet!
    //csc485b::a2::gpu::build_graph<<< 1, 1 >>>( g, d_edges, m );
    csc485b::a2::gpu::build_graph( g, d_edges, m, n );

    hipDeviceSynchronize();
    auto const reachability_start = std::chrono::high_resolution_clock::now();

    // neither does this!
    csc485b::a2::gpu::two_hop_reachability<<< dim3{n/32, n/32, n/32}, dim3{32, 32, 1} >>>( g );

    hipDeviceSynchronize();
    auto const end = std::chrono::high_resolution_clock::now();

    std::cout << "Build time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(reachability_start - build_start).count()
              << " us"
              << std::endl;


    std::cout << "Reachability time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
              << " us"
              << std::endl;
}

/**
 * Allocates space for a dense graph and then runs the test code on it.
 * Note that res is a float* so that it can be used with BLAS libraries
 */
void run_dense( csc485b::a2::edge_t const * d_edges, std::size_t n, std::size_t m,  float* res)
{
    using namespace csc485b;

    // allocate device DenseGraph
    a2::node_t * d_matrix, *d_dest;
    hipMalloc( (void**)&d_matrix, sizeof( a2::node_t ) * n * n );
    hipMalloc( (void**)&d_dest, sizeof( a2::node_t ) * n * n );
    a2::DenseGraph dg{ n, d_matrix, d_dest };
    a2::DenseGraph *d_dg;
    hipMalloc( (void**)&d_dg, sizeof( a2::DenseGraph ) );
    hipMemcpy( d_dg, &dg, sizeof( a2::DenseGraph ), hipMemcpyHostToDevice );

    run( d_dg, d_edges, m, n );

    // check output?
    std::vector< a2::node_t > host_matrix( dg.matrix_size() );
    std::vector< a2::node_t > host_dest( dg.matrix_size() );
    a2::DenseGraph dg_res{ n, host_matrix.data(), host_dest.data() };
    hipMemcpy( dg_res.adjacencyMatrix, dg.adjacencyMatrix, sizeof( a2::node_t ) * dg.matrix_size(), hipMemcpyDeviceToHost );
    hipMemcpy( dg_res.dest, dg.dest, sizeof( a2::node_t ) * dg.matrix_size(), hipMemcpyDeviceToHost );
    for (int idx = 0; idx < n; idx++)
    {
        //std::cout << idx << ": ";
        for (int jdx = 0; jdx < n; jdx++)
        {
            //std::cout << dg_res.dest[idx*n + jdx] << " ";
        }
        //std::cout << "\n";
    }

    bool check = true;
    for (int idx = 0; idx < n; idx++)
    {
        for (int jdx = 0; jdx < n; jdx++)
        {
            if (dg_res.dest[idx*n + jdx]*1.0 != res[idx*n + jdx]){
                check = false;
                break;
            }
        }
    }

    std::cout << "Correct output: " << check << "\n";

    // clean up
    hipFree( d_matrix );
}

/**
 * Allocates space for a sparse graph and then runs the test code on it.
 */
void run_sparse( csc485b::a2::edge_t const * d_edges, std::size_t n, std::size_t m )
{
    using namespace csc485b;

    // allocate device SparseGraph
    a2::node_t * d_offsets, * d_neighbours;
    hipMalloc( (void**)&d_offsets,    sizeof( a2::node_t ) * (n+1) );
    hipMalloc( (void**)&d_neighbours, sizeof( a2::node_t ) * m );
    a2::SparseGraph sg{n, m, d_offsets, d_neighbours };
    a2::SparseGraph *d_sg;
    hipMalloc( (void**)&d_sg, sizeof( a2::SparseGraph ) );
    hipMemcpy( d_sg, &sg, sizeof( a2::SparseGraph ), hipMemcpyHostToDevice );
    //a2::SparseGraph d_sg{ n, m, d_offsets, d_neighbours };

    run( d_sg, d_edges, m, n );

    // check output
    a2::SparseGraph *sg_res;
    a2::node_t *offsets, *neighbours;
    sg_res = (a2::SparseGraph*)malloc(sizeof( a2::SparseGraph));
    offsets = (a2::node_t*)malloc(sizeof( a2::node_t) * n);
    neighbours = (a2::node_t*)malloc(sizeof( a2::node_t) * m);
    hipMemcpy( sg_res, d_sg, sizeof( a2::SparseGraph ), hipMemcpyDeviceToHost );
    hipMemcpy( offsets, sg_res->neighbours_start_at, sizeof( a2::node_t ) * (n+1), hipMemcpyDeviceToHost );
    hipMemcpy( neighbours, d_neighbours, sizeof( a2::node_t ) * m, hipMemcpyDeviceToHost );

    std::cout << "m: " << sg_res->m << " n: " << sg_res->n << "\n";

    for (int idx = 0; idx < n+1; idx++)
    {
        std::cout << offsets[idx] << " ";
    }
    std::cout << "\n";

    for (int idx = 0; idx < m; idx++)
    {
        std::cout << neighbours[idx] << " ";
    }
    std::cout << "\n";

    // clean up
    hipFree( d_neighbours );
    hipFree( d_offsets );
    hipFree( d_sg );
    free(offsets);
    free(neighbours);
}

void matmul(float *mat, float *res, std::size_t n)
{
    for (std::size_t idx = 0; idx < n; idx++)
        for (std::size_t jdx = 0; jdx < n; jdx++)
            for (std::size_t kdx = 0; kdx < n; kdx++)
               res[idx*n + jdx] += mat[idx*n + kdx] * mat[kdx*n + jdx];
}

void print_matrix(float *matrix, std::size_t n){
      for (int row = 0; row < n; ++row){
        for (int col = 0; col < n; ++col){
            std::cout << matrix[(row * n) + col] << " ";
        }
        std::cout << "\n";
    }
}

void clamp(float *mat, std::size_t n)
{
    for (std::size_t idx = 0; idx < n; idx++)
    {
        for (std::size_t jdx = 0; jdx < n; jdx++)
        {
            if (idx == jdx || !mat[idx*n + jdx])
                mat[idx*n + jdx] = 0.0;
            else
                mat[idx*n + jdx] = 1.0;
        }
    }
}

int main()
{
    using namespace csc485b;
    
    // Create input
    std::size_t constexpr n = 4096;
    std::size_t constexpr expected_degree = n >> 2;

    a2::edge_list_t const graph = a2::generate_graph( n, n * expected_degree );
    std::size_t const m = graph.size();

    std::size_t padded_n = get_padded_sz(n);

    // lazily echo out input graph
    /*
    for( auto const& e : graph )
    {
        std::cout << "(" << e.x << "," << e.y << ") ";
    }
    std::cout << "\n";
    */

    // need to malloc since the matrix will exceed default stack size when n >= 1024
    float *matrix, *res;
    matrix = (float*) malloc(sizeof(float) * padded_n * padded_n);
    res = (float*) malloc(sizeof(float) * padded_n * padded_n);

    for (std::size_t idx = 0; idx < n*n; idx++)
    {
        matrix[idx] = 0.0;
        res[idx] = 0.0;
    }

    for ( auto const& e : graph ) {
        matrix[(e.x*padded_n) + e.y] = 1.0;
    }

    //print_matrix(matrix, padded_n);

    // OpenBLAS implementation
    auto const reachability_start = std::chrono::high_resolution_clock::now();

#ifdef NO_OPENBLAS
    // naive n^3 implementation
    matmul(matrix, res, padded_n);
#else
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, padded_n, padded_n, padded_n, 1.0,
                matrix, padded_n, matrix, padded_n, 1.0, res, padded_n);
#endif

    clamp(res, padded_n);
    auto const end = std::chrono::high_resolution_clock::now();

    std::cout << "Reachability time (CPU): "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
              << " us"
              << std::endl;

    //std::cout << "Correct output\n";
    //print_matrix(res, padded_n);

    // allocate and memcpy input to device
    a2::edge_t * d_edges;
    hipMalloc( (void**)&d_edges, sizeof( a2::edge_t ) * m );
    hipMemcpyAsync( d_edges, graph.data(), sizeof( a2::edge_t ) * m, hipMemcpyHostToDevice );

    // run your code!
    //run_sparse( d_edges, padded_n, m );
    //for (int idx = 0; idx < 10; idx++ )
        run_dense ( d_edges, padded_n, m, res );

    free(res);
    free(matrix);
    return EXIT_SUCCESS;
}
