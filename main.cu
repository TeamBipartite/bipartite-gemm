#include <chrono>   // for timing
#include <iostream> // std::cout, std::endl
#include <iterator> // std::ostream_iterator
#include <cstdlib> // EXIT_SUCCESS
#include <vector>

#ifndef NO_OPENBLAS
#include <cblas.h>
#endif

#include "utils.h"
#include "GEMM.h"

using namespace csc485b::a4;

/** 
 * main
 * @brief Performs GEMM on GPU CUDA Cores and Tensor Cores.
 */
int main(int argc, char **argv)
{   
    bool print_result = false;

    if (argc > 1 && !strncmp(argv[1], "-p", 3)) print_result = true;

    constexpr int n =  utils::get_padded_sz(16, 16);
    constexpr int max_element = 10;


    
    /*
    *******************************
    * Prepare Matrices
    *******************************
    */
    const std::vector<uint32_t> matrix_a = utils::generate_matrix<uint32_t>( max_element, n*n );
    uint32_t* d_matrix_a;

    const std::vector<uint32_t> matrix_b = utils::generate_matrix<uint32_t>( max_element, n*n );
    uint32_t* d_matrix_b;


    std::vector<uint32_t> matrix_c(n*n, 0);
    uint32_t* d_matrix_c;

    // Copy data to device
    utils::allocate_device_space( &d_matrix_a, &d_matrix_b, &d_matrix_c, n*n );

    // Copy contents of matrix_a and matrix_b to device
    hipMemcpy( d_matrix_a, matrix_a.data(), sizeof(uint32_t) * matrix_a.size(), hipMemcpyHostToDevice );
    hipMemcpy( d_matrix_b, matrix_b.data(), sizeof(uint32_t) * matrix_b.size(), hipMemcpyHostToDevice );

    // Set contents of matrix_c to zero on device
    hipMemset(d_matrix_c, 0x0, sizeof(half) * matrix_c.size() );

    /*
    *******************************
    * CUDA Core GEMM Implementation
    *******************************
    */


    
    // Perform GEMM of matrix a x b, storing the result in matrix c
    uint32_t block_dim_sz = (uint32_t)(n / 32);
    
    auto const cuda_core_gemm_start = std::chrono::high_resolution_clock::now();
    cudacores::matrix_mult<<< dim3{block_dim_sz, block_dim_sz, block_dim_sz}, dim3{32, 32, 1} >>>(d_matrix_a, d_matrix_b, d_matrix_c, n);
    hipDeviceSynchronize();
    auto const cuda_core_gemm_end = std::chrono::high_resolution_clock::now();

    // Copy results back to host
    hipMemcpy( matrix_c.data(), d_matrix_c, sizeof(uint32_t) * matrix_c.size(), hipMemcpyDeviceToHost );

    // Get results
    std::cout << "------CUDA Core GEMM Implementation------" << std::endl;
    #ifdef NO_OPENBLAS
        std::vector<uint32_t> matrix_c_expected = utils::naive_cpu_matmul( matrix_a, matrix_b, n );
        utils::print_matrix<uint32_t>( matrix_c_expected, n, print_result );
        utils::print_matrix<uint32_t>( matrix_c, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<uint32_t>(matrix_c, matrix_c_expected) << std::endl;
    #else
        const std::vector<uint32_t> matrix_c_float( matrix_c.begin(), matrix_c.end() );
        const std::vector<uint32_t> matrix_c_expected = utils::cblas_cpu_matmul<uint32_t,uint32_t>(matrix_a, matrix_b, n );
        utils::print_matrix<uint32_t>( matrix_c_expected, n, print_result );
        utils::print_matrix<uint32_t>( matrix_c_float, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<uint32_t>(matrix_c_float, matrix_c_expected) << std::endl;
    #endif
    std::cout << "CUDA Core GEMM Time: " 
              << std::chrono::duration_cast<std::chrono::microseconds>(cuda_core_gemm_end - cuda_core_gemm_start).count()
              << " us"
              << std::endl;
    

    /*
    *********************************
    * Tensor Core GEMM Implementation
    *********************************
    */

    /*
    const std::vector<half> matrix_a_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_a_fp16;

    const std::vector<half> matrix_b_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_b_fp16;

    const std::vector<half> matrix_c_fp16(n*n, 0);
    half* d_matrix_c_fp16;


    // Copy data to device
    utils::allocate_device_space( &d_matrix_a_fp16, &d_matrix_b_fp16, &d_matrix_c_fp16, n*n );

    // Copy contents of matrix_a and matrix_b to device
    hipMemcpy( d_matrix_a_fp16, matrix_a_fp16.data(), sizeof(half) * matrix_a_fp16.size(), hipMemcpyHostToDevice );
    hipMemcpy( d_matrix_b_fp16, matrix_b_fp16.data(), sizeof(half) * matrix_b_fp16.size(), hipMemcpyHostToDevice );

    // Set contents of matrix_c to zero on device
    hipMemset(d_matrix_c_fp16, 0x0, sizeof(half) * matrix_c_fp16.size() );


    std::cout << "------Tensor Core FP16 GEMM Implementation------" << std::endl;
    auto const tensor_core_gemm_start = std::chrono::high_resolution_clock::now();
    tensorcores::half_gemm<<< 1, dim3{128, 4, 1} >>>(d_matrix_a_fp16, d_matrix_b_fp16, d_matrix_c_fp16, n);
    hipMemcpy( matrix_c_fp16.data(), d_matrix_c_fp16, sizeof(half) * matrix_c_fp16.size(), hipMemcpyDeviceToHost );
    auto const tensor_core_gemm_end = std::chrono::high_resolution_clock::now();

    const std::vector<half> matrix_c_expected_fp16 = utils::cblas_cpu_matmul<half,half>(matrix_a_fp16, matrix_b_fp16, n );
    utils::print_matrix<half>( matrix_c_expected_fp16, n, print_result );
    utils::print_matrix<half>( matrix_c_fp16, n, print_result );
    std::cout << "Correct Output:" << utils::matrices_equal<half>(matrix_c_fp16, matrix_c_expected_fp16) << std::endl;

    */

    const std::vector<half> matrix_a_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_a_fp16;

    const std::vector<half> matrix_b_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_b_fp16;
    
    std::vector<half> matrix_c_fp32(n*n, 0);
    half* d_matrix_c_fp32;

    // Copy data to device
    utils::allocate_device_space( &d_matrix_a_fp16, &d_matrix_b_fp16, &d_matrix_c_fp32, n*n );

    // Copy contents of matrix_a and matrix_b to device
    hipMemcpy( d_matrix_a_fp16, matrix_a_fp16.data(), sizeof(half) * matrix_a_fp16.size(), hipMemcpyHostToDevice );
    hipMemcpy( d_matrix_b_fp16, matrix_b_fp16.data(), sizeof(half) * matrix_b_fp16.size(), hipMemcpyHostToDevice );

    // Set contents of matrix_c to zero on device
    hipMemset(d_matrix_c, 0x0, sizeof(half) * matrix_c.size() );
    

    dim3 gridDim;
    dim3 blockDim;
    blockDim.x = 128;
    blockDim.y = 4;
    gridDim.x = (n + (16 * blockDim.x / 32 - 1)) / (16 * blockDim.x / 32);
    gridDim.y = (n + 16 * blockDim.y - 1) / (16 * blockDim.y);

    auto const tensor_core_gemm_start = std::chrono::high_resolution_clock::now();
    tensorcores::half_gemm<<< 1, dim3{128, 4, 1} >>>(d_matrix_a_fp16, d_matrix_b_fp16, d_matrix_c_fp32, n);
    hipDeviceSynchronize();
    auto const tensor_core_gemm_end = std::chrono::high_resolution_clock::now();

    hipMemcpy( matrix_c_fp32.data(), d_matrix_c_fp32, sizeof(half) * matrix_c_fp32.size(), hipMemcpyDeviceToHost );


    // Get results
    std::cout << "------Tensor Core GEMM Implementation------" << std::endl;
    #ifdef NO_OPENBLAS
        std::vector<float> matrix_c_expected_fp32 = utils::naive_cpu_matmul<half, float>( matrix_a_fp16, matrix_b_fp16, n );
        utils::print_matrix<float>( matrix_c_expected_fp32, n, print_result );
        utils::print_matrix<float>( matrix_c_fp32, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<float>(matrix_c_fp32, matrix_c_expected_fp32) << std::endl;
    #else
        const std::vector<half> matrix_c_expected_fp32 = utils::cblas_cpu_matmul<half, half>(matrix_a_fp16, matrix_b_fp16, n );
        utils::print_matrix<half>( matrix_c_expected_fp32, n, print_result );
        utils::print_matrix<half>( matrix_c_fp32, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<half>(matrix_c_fp32, matrix_c_expected_fp32) << std::endl;
    #endif
    std::cout << "Tensor Core GEMM Time: " 
              << std::chrono::duration_cast<std::chrono::microseconds>(tensor_core_gemm_end - tensor_core_gemm_start).count()
              << " us"
              << std::endl;

    /*
    const std::vector<half> matrix_a_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_a_fp16;

    const std::vector<half> matrix_b_fp16 = utils::generate_matrix<half>( max_element, n*n );
    half* d_matrix_b_fp16;
    
    std::vector<float> matrix_c_fp32(n*n, 0);
    float* d_matrix_c_fp32;

    // Copy data to device
    utils::allocate_device_space( &d_matrix_a_fp16, &d_matrix_b_fp16, &d_matrix_c_fp32, n*n );

    // Copy contents of matrix_a and matrix_b to device
    hipMemcpy( d_matrix_a_fp16, matrix_a_fp16.data(), sizeof(half) * matrix_a_fp16.size(), hipMemcpyHostToDevice );
    hipMemcpy( d_matrix_b_fp16, matrix_b_fp16.data(), sizeof(half) * matrix_b_fp16.size(), hipMemcpyHostToDevice );

    // Set contents of matrix_c to zero on device
    hipMemset(d_matrix_c, 0x0, sizeof(float) * matrix_c.size() );
    

    dim3 gridDim;
    dim3 blockDim;
    blockDim.x = 128;
    blockDim.y = 4;
    gridDim.x = (n + (16 * blockDim.x / 32 - 1)) / (16 * blockDim.x / 32);
    gridDim.y = (n + 16 * blockDim.y - 1) / (16 * blockDim.y);

    auto const tensor_core_gemm_start = std::chrono::high_resolution_clock::now();
    tensorcores::fp32_wmma_gemm<<< gridDim, blockDim >>>(d_matrix_a_fp16, d_matrix_b_fp16, d_matrix_c_fp32, n);
    hipDeviceSynchronize();
    auto const tensor_core_gemm_end = std::chrono::high_resolution_clock::now();

    hipMemcpy( matrix_c_fp32.data(), d_matrix_c_fp32, sizeof(float) * matrix_c_fp32.size(), hipMemcpyDeviceToHost );


    // Get results
    std::cout << "------Tensor Core GEMM Implementation------" << std::endl;
    #ifdef NO_OPENBLAS
        std::vector<float> matrix_c_expected_fp32 = utils::naive_cpu_matmul<half, float>( matrix_a_fp16, matrix_b_fp16, n );
        utils::print_matrix<float>( matrix_c_expected_fp32, n, print_result );
        utils::print_matrix<float>( matrix_c_fp32, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<float>(matrix_c_fp32, matrix_c_expected_fp32) << std::endl;
    #else
        const std::vector<float> matrix_c_expected_fp32 = utils::cblas_cpu_matmul<half, float>(matrix_a_fp16, matrix_b_fp16, n );
        utils::print_matrix<float>( matrix_c_expected_fp32, n, print_result );
        utils::print_matrix<float>( matrix_c_fp32, n, print_result );
        std::cout << "Correct Output:" << utils::matrices_equal<float>(matrix_c_fp32, matrix_c_expected_fp32) << std::endl;
    #endif
    std::cout << "Tensor Core GEMM Time: " 
              << std::chrono::duration_cast<std::chrono::microseconds>(tensor_core_gemm_end - tensor_core_gemm_start).count()
              << " us"
              << std::endl;
    */


    // Cleanup
    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    hipFree(d_matrix_c);
    hipFree(d_matrix_a_fp16);
    hipFree(d_matrix_b_fp16);
    //hipFree(d_matrix_c_fp32);

    return EXIT_SUCCESS;
}
