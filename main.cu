/**
 * Driver for the benchmark comparison. Generates random data,
 * runs the CPU baseline, and then runs your code.
 */

#include <chrono>   // for timing
#include <iostream> // std::cout, std::endl
#include <iterator> // std::ostream_iterator
#include <vector>

#ifndef NO_OPENBLAS
#include <cblas.h>
#endif

#include "dense_graph.h"
#include "sparse_graph.h"

#include "data_generator.h"
#include "data_types.h"

/** 
 * Pads the given n to the least multiple of 32 not less than n
 */
constexpr std::size_t get_padded_sz(std::size_t n)
{
    return n%32 ? n + (32 - n%32) : n;
}

/**
 * Runs timing tests on a CUDA graph implementation.
 * Consists of independently constructing the graph and then
 * modifying it to its two-hop neighbourhood.
 */
template < typename DeviceGraph >
void run( DeviceGraph *g, csc485b::a2::edge_t const * d_edges, std::size_t m, std::size_t n )
{
    hipDeviceSynchronize();
    auto const build_start = std::chrono::high_resolution_clock::now();

    // this code doesn't work yet!
    //csc485b::a2::gpu::build_graph<<< 1, 1 >>>( g, d_edges, m );
    csc485b::a2::gpu::build_graph( g, d_edges, m, n );

    hipDeviceSynchronize();
    auto const reachability_start = std::chrono::high_resolution_clock::now();

    // neither does this!
    //csc485b::a2::gpu::two_hop_reachability( g, n, get_padded_sz(m) );

    hipDeviceSynchronize();
    auto const end = std::chrono::high_resolution_clock::now();

    std::cout << "Build time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(reachability_start - build_start).count()
              << " us"
              << std::endl;


    std::cout << "Reachability time: "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
              << " us"
              << std::endl;
}

/**
 * Allocates space for a dense graph and then runs the test code on it.
 * Note that res is a float* so that it can be used with BLAS libraries
 */
void run_dense( csc485b::a2::edge_t const * d_edges, std::size_t n, std::size_t m,  float* res)
{
    using namespace csc485b;

    // allocate device DenseGraph
    a2::node_t * d_matrix, *d_dest;
    hipMalloc( (void**)&d_matrix, sizeof( a2::node_t ) * n * n );
    hipMalloc( (void**)&d_dest, sizeof( a2::node_t ) * n * n );
    a2::DenseGraph dg{ n, d_matrix, d_dest };
    a2::DenseGraph *d_dg;
    hipMalloc( (void**)&d_dg, sizeof( a2::DenseGraph ) );
    hipMemcpy( d_dg, &dg, sizeof( a2::DenseGraph ), hipMemcpyHostToDevice );

    run( d_dg, d_edges, m, n );

    // check output?
    std::vector< a2::node_t > host_matrix( dg.matrix_size() );
    std::vector< a2::node_t > host_dest( dg.matrix_size() );
    a2::DenseGraph dg_res{ n, host_matrix.data(), host_dest.data() };
    hipMemcpy( dg_res.adjacencyMatrix, dg.adjacencyMatrix, sizeof( a2::node_t ) * dg.matrix_size(), hipMemcpyDeviceToHost );
    hipMemcpy( dg_res.dest, dg.dest, sizeof( a2::node_t ) * dg.matrix_size(), hipMemcpyDeviceToHost );
    for (int idx = 0; idx < n; idx++)
    {
        std::cout << idx << ": ";
        for (int jdx = 0; jdx < n; jdx++)
        {
            std::cout << dg_res.dest[idx*n + jdx] << " ";
        }
        std::cout << "\n";
    }

    bool check = true;
    for (int idx = 0; idx < n; idx++)
    {
        for (int jdx = 0; jdx < n; jdx++)
        {
            if (dg_res.dest[idx*n + jdx]*1.0 != res[idx*n + jdx]){
                check = false;
                break;
            }
        }
    }

    std::cout << "Correct output: " << check << "\n";

    // clean up
    hipFree( d_matrix );
}

/**
 * Allocates space for a sparse graph and then runs the test code on it.
 */
void run_sparse( csc485b::a2::edge_t const * d_edges, std::size_t n, std::size_t m, csc485b::a2::SparseGraph *res )
{
    using namespace csc485b;

    // allocate device SparseGraph
    a2::node_t * d_offsets, * d_neighbours;
    hipMalloc( (void**)&d_offsets,    sizeof( a2::node_t ) * (n+1) );
    hipMalloc( (void**)&d_neighbours, sizeof( a2::node_t ) * m );
    a2::SparseGraph sg{n, m, d_offsets, d_neighbours };
    a2::SparseGraph *d_sg;
    hipMalloc( (void**)&d_sg, sizeof( a2::SparseGraph ) );
    hipMemcpy( d_sg, &sg, sizeof( a2::SparseGraph ), hipMemcpyHostToDevice );
    //a2::SparseGraph d_sg{ n, m, d_offsets, d_neighbours };

    run( d_sg, d_edges, m, n );

    // check output
    a2::SparseGraph *sg_res;
    a2::node_t *offsets, *neighbours;
    sg_res = (a2::SparseGraph*)malloc(sizeof( a2::SparseGraph));
    offsets = (a2::node_t*)malloc(sizeof( a2::node_t) * n);
    neighbours = (a2::node_t*)malloc(sizeof( a2::node_t) * m);
    hipMemcpy( sg_res, d_sg, sizeof( a2::SparseGraph ), hipMemcpyDeviceToHost );
    hipMemcpy( offsets, sg_res->neighbours_start_at, sizeof( a2::node_t ) * (n+1), hipMemcpyDeviceToHost );
    hipMemcpy( neighbours, d_neighbours, sizeof( a2::node_t ) * m, hipMemcpyDeviceToHost );

    std::cout << "m: " << sg_res->m << " n: " << sg_res->n << "\n";
    int check = 1; 

    for (int idx = 0; idx < n+1; idx++)
    {
        if (offsets[idx] != res->neighbours_start_at[idx]) check = 0; 
        std::cout << offsets[idx] << " ";
    }
    std::cout << "\n";

    for (int idx = 0; idx < m; idx++)
    {
        if (neighbours[idx] != res->neighbours[idx]) check = 0; 
        std::cout << neighbours[idx] << " ";
    }
    std::cout << "\nCorrect output: " << check << "\n";

    // clean up
    hipFree( d_neighbours );
    hipFree( d_offsets );
    hipFree( d_sg );
    free(offsets);
    free(neighbours);
}

void matmul(float *mat, float *res, std::size_t n)
{
    for (std::size_t idx = 0; idx < n; idx++)
        for (std::size_t jdx = 0; jdx < n; jdx++)
            for (std::size_t kdx = 0; kdx < n; kdx++)
               res[idx*n + jdx] += mat[idx*n + kdx] * mat[kdx*n + jdx];
}

void print_matrix(float *matrix, std::size_t n){
      for (int row = 0; row < n; ++row){
        for (int col = 0; col < n; ++col){
            std::cout << matrix[(row * n) + col] << " ";
        }
        std::cout << "\n";
    }
}

void clamp(float *mat, std::size_t n)
{
    for (std::size_t idx = 0; idx < n; idx++)
    {
        for (std::size_t jdx = 0; jdx < n; jdx++)
        {
            if (idx == jdx || !mat[idx*n + jdx])
                mat[idx*n + jdx] = 0.0;
            else
                mat[idx*n + jdx] = 1.0;
        }
    }
}

int main()
{
    using namespace csc485b;
    
    // Create input
    std::size_t constexpr n = 32;
    std::size_t constexpr expected_degree = n >> 2;

    a2::edge_list_t const graph = a2::generate_graph( n, n * expected_degree );
    std::size_t const m = graph.size();

    std::size_t padded_n = get_padded_sz(n);

    // lazily echo out input graph
    
    //for( auto const& e : graph )
    //{
     //   std::cout << "(" << e.x << "," << e.y << ") ";
    //}
    //std::cout << "\n";
    

    // need to malloc since the matrix will exceed default stack size when n >= 1024
    float *matrix, *res;
    matrix = (float*) malloc(sizeof(float) * padded_n * padded_n);
    res = (float*) malloc(sizeof(float) * padded_n * padded_n);

    for (std::size_t idx = 0; idx < n*n; idx++)
    {
        matrix[idx] = 0.0;
        res[idx] = 0.0;
    }

    for ( auto const& e : graph ) {
        matrix[(e.x*padded_n) + e.y] = 1.0;
    }

    //print_matrix(matrix, padded_n);

    auto const reachability_start = std::chrono::high_resolution_clock::now();

#ifdef NO_OPENBLAS
    // naive n^3 implementation
    matmul(matrix, res, padded_n);
#else
    // OpenBLAS implementation
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, padded_n, padded_n, padded_n, 1.0,
                matrix, padded_n, matrix, padded_n, 1.0, res, padded_n);
#endif

    clamp(res, padded_n);

    a2::SparseGraph res_csr{n, m};
    res_csr.neighbours = (a2::node_t*) malloc(sizeof(a2::node_t) * m);
    res_csr.neighbours_start_at = (a2::node_t*) malloc(sizeof(a2::node_t) * padded_n+1);

    std::size_t cur_idx = 0;
    for (int idx = 0; idx < padded_n+1; idx++)
    {
        res_csr.neighbours_start_at[idx] = cur_idx;
        for (int jdx = 0; jdx < n; jdx++)
        {
            if (matrix[idx*padded_n + jdx] > 0)
            {
                res_csr.neighbours[cur_idx++] = jdx;
            }
        }
    }

    for (int idx = 0; idx < padded_n+1; idx++)
    {
        std::cout << res_csr.neighbours_start_at[idx] << " ";
    }
    std::cout << "\n"; 

    for (int idx = 0; idx < m; idx++)
    {
        std::cout << res_csr.neighbours[idx] << " ";
    }

    std::cout << "\n";

    auto const end = std::chrono::high_resolution_clock::now();

    std::cout << "Reachability time (CPU): "
              << std::chrono::duration_cast<std::chrono::microseconds>(end - reachability_start).count()
              << " us"
              << std::endl;

    //std::cout << "Correct output\n";
    //print_matrix(res, padded_n);

    // allocate and memcpy input to device
    a2::edge_t * d_edges;
    hipMalloc( (void**)&d_edges, sizeof( a2::edge_t ) * m );
    hipMemcpyAsync( d_edges, graph.data(), sizeof( a2::edge_t ) * m, hipMemcpyHostToDevice );

    // run your code!
    run_sparse( d_edges, padded_n, m, &res_csr );
    //for (int idx = 0; idx < 10; idx++ )
    //    run_dense ( d_edges, padded_n, m, res );

    free(res);
    free(matrix);
    return EXIT_SUCCESS;
}
