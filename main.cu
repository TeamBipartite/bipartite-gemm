#include <chrono>   // for timing
#include <iostream> // std::cout, std::endl
#include <iterator> // std::ostream_iterator
#include <cstdlib> // EXIT_SUCCESS
#include <vector>

#ifndef NO_OPENBLAS
#include <cblas.h>
#endif

#include "gemm_experiment.h"
#include "GEMM.h"

using namespace csc485b::a4;

/** 
 * main
 * @brief Performs GEMM on GPU CUDA Cores and Tensor Cores.
 */
int main(int argc, char **argv)
{   
    bool print_result = false;

    if (argc > 1 && !strncmp(argv[1], "-p", 3)) print_result = true;

    constexpr int original_n  = 8192;
    constexpr int multiple    = WMMA_M * 4;
    constexpr int max_element = 40;

    // Create a random device
    std::random_device rd;
    // Generate a random seed to ensure all experiments use the same random values
    unsigned int seed = rd();

    /*
    ****************************
    * CUDA Core GEMM Experiment
    ****************************
    */
    
    GemmExperiment<uint32_t, uint32_t> cudaCoreExp{original_n, max_element, multiple, seed, print_result};
    cudaCoreExp.run_experiment( 
        [&cudaCoreExp] (uint32_t* a, uint32_t* b, uint32_t* c) {
            uint32_t block_dim_sz = (uint32_t)(cudaCoreExp.get_n() / WARP_SZ);
            const dim3 gridDim{block_dim_sz, block_dim_sz, block_dim_sz};
            const dim3 blockDim{WARP_SZ, WARP_SZ, 1};
            cudacores::matrix_mult<<< gridDim, blockDim >>>(a, b, c, cudaCoreExp.get_n());
            }, "CUDA Core GEMM Implementation" );

    /*
    **********************************
    * Tensor Core FP32 GEMM Experiment
    **********************************
    */
    
    GemmExperiment<half, float> tensorCoreExpFp32{original_n, max_element, multiple, seed, print_result};
    tensorCoreExpFp32.run_experiment(
        [&tensorCoreExpFp32] (half* a, half* b, float* c) {
            const dim3 blockDim { WARP_SZ * 4, 4, 1 };
            dim3 gridDim;
            gridDim.x = (tensorCoreExpFp32.get_n() + (WMMA_N * blockDim.x / WARP_SZ - 1)) / (WMMA_N * blockDim.x / WARP_SZ);
            gridDim.y = (tensorCoreExpFp32.get_n() + WMMA_M * blockDim.y - 1) / (WMMA_M * blockDim.y);
            tensorcores::gemm<half, float><<< gridDim, blockDim >>>(a, b, c, tensorCoreExpFp32.get_n());
            }, "Tensor Core GEMM FP32 Implementation" );

    /*
    **********************************
    * Tensor Core FP32 (multi-stream) GEMM Experiment
    **********************************
    */
    std::size_t superblock_sz = 128;
    GemmExperiment<half, float> tensorCoreExpFp32Streams{original_n, max_element, multiple, seed, print_result, superblock_sz};
    std::size_t padded_n = tensorCoreExpFp32Streams.get_n();
    tensorCoreExpFp32Streams.run_experiment_streams( 
        [&tensorCoreExpFp32Streams, padded_n, superblock_sz] (half *a, half *b, float *c, hipStream_t stream) {
            const dim3 blockDim { WARP_SZ * 4, 4, 1 };
            dim3 gridDim;
            gridDim.x = (padded_n + (WMMA_N * blockDim.x / WARP_SZ - 1)) / (WMMA_N * blockDim.x / WARP_SZ);
            gridDim.y = (superblock_sz + WMMA_M * blockDim.y - 1) / (WMMA_M * blockDim.y);
            tensorcores::gemm<half, float><<< gridDim, blockDim, 0, stream >>>(a, b, c, padded_n, superblock_sz);
            }, "Tensor Core GEMM FP32 (two streams) Implementation" );
    
    /*
    **********************************
    * Tensor Core FP16 GEMM Experiment
    **********************************
    */

    GemmExperiment<half, half> tensorCoreExpFp16{original_n, max_element, multiple, seed, print_result};
    padded_n = tensorCoreExpFp16.get_n();
    tensorCoreExpFp16.run_experiment( 
        [&tensorCoreExpFp16, padded_n] (half* a, half* b, half* c) {
            const dim3 blockDim { WARP_SZ * 4, 4, 1 };
            dim3 gridDim;
            gridDim.x = (padded_n + (WMMA_M * blockDim.x / WARP_SZ - 1)) / (WMMA_M * blockDim.x / WARP_SZ);
            gridDim.y = (padded_n + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
            tensorcores::gemm<half, half><<< gridDim, blockDim >>>(a, b, c, padded_n);
            }, "Tensor Core GEMM FP16 Implementation" );


    /*
    **********************************
    * Tensor Core FP16 (multi-stream) GEMM Experiment
    **********************************
    */

    superblock_sz = 128;
    GemmExperiment<half, half> tensorCoreExpFp16Streams{original_n, max_element, multiple, seed, print_result, superblock_sz};
    padded_n = tensorCoreExpFp16Streams.get_n();
    tensorCoreExpFp16Streams.run_experiment_streams( 
        [&tensorCoreExpFp16Streams, padded_n, superblock_sz] (half *a, half *b, half *c, hipStream_t stream) {
            const dim3 blockDim { WARP_SZ * 4, 4, 1 };
            dim3 gridDim;
            gridDim.x = (padded_n + (WMMA_N * blockDim.x / WARP_SZ - 1)) / (WMMA_N * blockDim.x / WARP_SZ);
            gridDim.y = (superblock_sz + WMMA_M * blockDim.y - 1) / (WMMA_M * blockDim.y);
            tensorcores::gemm<half, half><<< gridDim, blockDim, 0, stream >>>(a, b, c, padded_n, superblock_sz);
            }, "Tensor Core GEMM FP16 (two streams) Implementation" );

    /*
    **********************************
    * Tensor Core INT8 GEMM Experiment
    **********************************
    */

    GemmExperiment<unsigned char, int> tensorCoreExpInt8{original_n, max_element, multiple, seed, print_result};
    tensorCoreExpInt8.run_experiment(
        [&tensorCoreExpInt8] (unsigned char *a, unsigned char *b, int *c) {
            const dim3 blockDim { WARP_SZ * 4, 4, 1 };
            dim3 gridDim;
            gridDim.x = (tensorCoreExpInt8.get_n() + (WMMA_N * blockDim.x / WARP_SZ - 1)) / (WMMA_N * blockDim.x / WARP_SZ);
            gridDim.y = (tensorCoreExpInt8.get_n() + WMMA_M * blockDim.y - 1) / (WMMA_M * blockDim.y);
            tensorcores::gemm<unsigned char, int><<< gridDim, blockDim >>>(a, b, c, tensorCoreExpInt8.get_n());
            }, "Tensor Core GEMM INT8 Implementation" );

    return EXIT_SUCCESS;
}
